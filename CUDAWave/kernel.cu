#include "hip/hip_runtime.h"
#include <simple2d.h>

#include <hip/hip_runtime.h>
#include <iostream>
#include <string>

#define PI 3.14159265358979323846

S2D_Window* window;

float margins = 10;
float scaling = 50;
float weight = 2;
bool paused = false;
bool mousePressed = false;
float scrollSpeed = 0.05;

bool useCPU = false;

const double frequency = 10;

const double length = 1; // length in m, should be a whole number
const double ppm = 2048; // points per meter, should be a multiple of 1024
const double c = 1; // m/s

double dt = 0.00001;
int stepsPerFrame = 1000;

int nPoints = (int)(length * ppm);
double dx = length / nPoints;
double t = 0;

double* pos, * vel, * nPos;
double* d_pos, * d_vel, * d_nPos;
double* d_dx, * d_dt, * d_c;
int* d_N;

int NUM_THREADS = 1 << 10; // max (1024) threads per thread block
int NUM_BLOCKS = (nPoints + NUM_THREADS - 1) / NUM_THREADS;

int deviceId = -1;
bool GPUAllocated = false;

S2D_Text* txt = S2D_CreateText("LEMONMILK-Light.otf", "a", 15);

__global__ void timeStep(double* pos, double* vel, double* nPos, int* N, double* dx, double* dt, double* c, double input) {
	int TID = threadIdx.x + (blockIdx.x * blockDim.x);

	if (TID >= *N) {
		return;
	}

	if (TID == 0) {
		nPos[TID] = input;
		return;
	}

	double leftSlope = (pos[TID] - pos[TID - 1]) / (*dx);
	double rightSlope;
	if (TID == (*N) - 1) {
		rightSlope = leftSlope;
	}
	else {
		rightSlope = (pos[TID + 1] - pos[TID]) / (*dx);
	}

	double acc = pow((*c), 2.0) * (rightSlope - leftSlope) / (*dx);

	vel[TID] = vel[TID] + acc * (*dt);
	nPos[TID] = pos[TID] + vel[TID] * (*dt);
}

void timeStepCPU(int TID, double input) {
	if (TID >= nPoints) {
		return;
	}

	if (TID == 0) {
		nPos[TID] = input;
		return;
	}

	double leftSlope = (pos[TID] - pos[TID - 1]) / dx;
	double rightSlope;
	if (TID == (nPoints) - 1) {
		rightSlope = leftSlope;
	}
	else {
		rightSlope = (pos[TID + 1] - pos[TID]) / dx;
	}

	double acc = pow(c, 2.0) * (rightSlope - leftSlope) / dx;

	vel[TID] = vel[TID] + acc * dt;
	nPos[TID] = pos[TID] + vel[TID] * dt;
}

double wave(double time) {
	if (frequency * time > 20) return 0;
	return (-cos(2.0 * PI * frequency * time) + 1);
}

void sendToGPU() {
	if (useCPU) return;
	if (!GPUAllocated) {
		hipMalloc(&d_pos, nPoints * sizeof(double));
		hipMalloc(&d_vel, nPoints * sizeof(double));
		hipMalloc(&d_nPos, nPoints * sizeof(double));
		hipMalloc(&d_N, sizeof(int));
		hipMalloc(&d_dx, sizeof(double));
		hipMalloc(&d_dt, sizeof(double));
		hipMalloc(&d_c, sizeof(double));
		GPUAllocated = true;
	}

	hipMemcpy(d_pos, pos, nPoints * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_vel, pos, nPoints * sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_N, &nPoints, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_dx, &dx, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_dt, &dt, sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(d_c, &c, sizeof(double), hipMemcpyHostToDevice);
}

void swapPointers() {
	double* d_pos_temp, * pos_temp;

	// swap GPU pointers
	d_pos_temp = d_pos;
	d_pos = d_nPos;
	d_nPos = d_pos_temp;

	// swap CPU pointers
	pos_temp = pos;
	pos = nPos;
	nPos = pos_temp;
}

void reset() {
	for (int i = 0; i < nPoints; i++) {
		pos[i] = 0;
		vel[i] = 0;
		nPos[i] = 0;
	}
	t = 0;

	sendToGPU();
}

void setupVars() {
	pos = new double[nPoints];
	vel = new double[nPoints];
	nPos = new double[nPoints];
	reset();
}

std::string getDeviceName(int device) {
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	return deviceProp.name;
}

void selectDevice() {
	int deviceCount;
	auto result = hipGetDeviceCount(&deviceCount);
	if (deviceCount < 1 || result != hipSuccess) {
		std::cout << "No CUDA enabled GPU's detected. Using CPU." << std::endl;
		useCPU = true;
		return;
	}

	if (deviceCount == 1) {
		std::cout << "Only one device detected, using that one." << std::endl;
		deviceId = 0;
		return;
	}
	std::cout << deviceCount << " compatible devices detected, please pick one: " << std::endl;

	while (deviceId < 0 || deviceId > deviceCount - 1) {
		for (int dev = 0; dev < deviceCount; dev++) {
			std::cout << "(" << dev << ") " << getDeviceName(dev) << std::endl;
		}

		std::string input = "";
		std::cin >> input;

		try {
			deviceId = std::stoi(input, nullptr, 10);
		}
		catch (...) {
			deviceId = -1;
		}

		if(deviceId < 0 || deviceId > deviceCount - 1) {
			std::cout << "Please enter a valid option." << std::endl;
		}
	}

	hipSetDevice(deviceId);

}

void on_key(S2D_Event e) {
	if (e.type != S2D_KEY_DOWN) return;
	std::cout << e.key << " pressed" << std::endl;
	if (strcmp(e.key, "Space") == 0) {
		paused = !paused;
	}
	if (strcmp(e.key, "R") == 0) {
		reset();
	}
}

void on_mouse(S2D_Event e) {
	switch (e.type) {
	case S2D_MOUSE_SCROLL:
		int ddt = -scrollSpeed*(stepsPerFrame * e.delta_y + 1);
		if (ddt == 0) ddt = (e.delta_y < 0 ? 1 : -1);
		stepsPerFrame = stepsPerFrame + ddt;
		if (stepsPerFrame < 1) stepsPerFrame = 1;
		break;
	}
}

void update() {
	if (paused) {
		return;
	}

	int iteration = 0;
	while (iteration < stepsPerFrame) {
		double input = wave(t);
		if (useCPU) {
			for (int p = 0; p < nPoints; p++) {
				timeStepCPU(p, input);
			}
		}
		else {
			timeStep << <NUM_BLOCKS, NUM_THREADS >> > (d_pos, d_vel, d_nPos, d_N, d_dx, d_dt, d_c, input);
		}
		swapPointers();
		t += dt;
		iteration++;
	}
	if(!useCPU)
		hipMemcpy(pos, d_pos, nPoints * sizeof(double), hipMemcpyDeviceToHost);
}

void drawText(std::string text, float x, float y) {
	S2D_SetText(txt, text.c_str());
	txt->x = x;
	txt->y = y;
	S2D_DrawText(txt);
}

int currentLine = 0;
void printLine(std::string text) {
	drawText(text, 5, 5 + 15 * currentLine);
	currentLine++;
}

void render() {
	currentLine = 0;
	float y0 = window->viewport.height / 2;
	float x0 = margins;
	float spacing = (window->viewport.width - 2.0 * margins) / nPoints;
	for (int i = 0; i < nPoints - 1; i++) {
		S2D_DrawLine(
			x0 + spacing * i, y0 - scaling * pos[i],
			x0 + spacing * (i + 1), y0 - scaling * pos[i + 1],
			weight,
			1, 1, 1, 1,
			1, 1, 1, 1,
			1, 1, 1, 1,
			1, 1, 1, 1
		);
	}

	float timePerFrame = dt * stepsPerFrame;
	float simSpeed = timePerFrame * window->fps;

	printLine(paused ? "Paused" : "Running");
	printLine("Time: " + std::to_string(t));
	printLine("Steps per frame: " + std::to_string(stepsPerFrame));
	printLine("Step size: " + std::to_string(dt));
	printLine("Device: " + std::string(useCPU ? "CPU" : getDeviceName(deviceId)));
	printLine("FPS: " + std::to_string(window->fps));
	printLine("Simulation speed: " + std::to_string(simSpeed*100) + "% of realtime");
	printLine("");
	printLine("Controls:");
	printLine("Space: Pause simulation");
	printLine("R: Reset simulation");
	printLine("Scroll: Change simulation speed");
}

int main(int argc, char* argv[]) {
	selectDevice();
	setupVars();

	std::cout
		<< "Device: " << getDeviceName(deviceId) << std::endl
		<< "Blocks: " << NUM_BLOCKS << std::endl
		<< "Threads per block: " << NUM_THREADS << std::endl
		<< "Total threads: " << NUM_THREADS * NUM_BLOCKS << std::endl
		<< "Points: " << nPoints << std::endl;

	int inactiveThreads = NUM_THREADS * NUM_BLOCKS - nPoints;
	if (inactiveThreads) {
		std::cout << "WARNING: " << inactiveThreads << " inactive threads detected. Concider increasing nPoints to an integer multiple of 1024" << std::endl;
	}

	window = S2D_CreateWindow(
		"WaveSim",
		1280, 720,
		update, render,
		S2D_RESIZABLE
	);

	window->on_key = on_key;
	window->on_mouse = on_mouse;

	S2D_Show(window);

	return 0;

}